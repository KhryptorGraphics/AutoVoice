#include "hip/hip_runtime.h"
#include "kernel_utils.cuh"
#include "fft_ops.cuh"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_cooperative_groups.h>
#include <>
#include <torch/extension.h>
#include <algorithm>

using namespace cooperative_groups;

// Pitch detection kernel using autocorrelation (simplified YIN)
__global__ void pitch_detection_kernel(float *audio, float *pitch, int n_samples, int frame_length, int hop_length, float fmin, float fmax, float threshold, float sample_rate) {
    int frame_idx = blockIdx.x;
    int tid = threadIdx.x;
    
    extern __shared__ float shared_audio[];
    int frame_start = frame_idx * hop_length;

    // Load frame into shared memory with bounds check
    bool in_bounds = (frame_start + tid < n_samples) && (tid < frame_length);
    if (in_bounds) {
        shared_audio[tid] = audio[frame_start + tid];
    } else if (tid < frame_length) {
        shared_audio[tid] = 0.0f;  // Zero padding
    }
    __syncthreads();
    
    int tau_min = (int)(sample_rate / fmax);
    int tau_max = (int)(sample_rate / fmin);

    float diff_mean = 0.0f;
    float cum = 0.0f;
    float bottom = 0.0f;

    for (int tau = tau_min; tau <= tau_max; ++tau) {
        float acf = 0.0f;

        // Compute autocorrelation
        for (int j = 0; j < frame_length - tau; j += blockDim.x) {
            if (tid + j < frame_length - tau) {
                float diff = shared_audio[tid + j] - shared_audio[tid + j + tau];
                acf += diff * diff;
            }
        }
        acf = warp_reduce_sum(acf);

        if (frame_idx == 0 && tau == tau_min) {
            diff_mean = acf;
        }

        float diff = acf / frame_length;
        cum += diff;
        if (tau > tau_min) {
            bottom += diff * diff;
        }

        float running_measure = cum / (float)tau;
        float normalized = diff / diff_mean;

        if (running_measure < threshold) {
            pitch[frame_idx] = sample_rate / (float)tau;
            return;
        }
    }

    pitch[frame_idx] = 0.0f; // No pitch detected
}

// Voice Activity Detection kernel
__global__ void vad_kernel(float *audio, float *vad, int n_samples, int frame_length, int hop_length, float threshold) {
    int frame_idx = blockIdx.x;
    int tid = threadIdx.x;

    int frame_start = frame_idx * hop_length;

    // Compute frame energy
    extern __shared__ float shared_energy[];
    bool in_bounds = (frame_start + tid) < n_samples;
    float sample = in_bounds ? audio[frame_start + tid] : 0.0f;
    shared_energy[tid] = in_bounds ? sample * sample : 0.0f;

    __syncthreads();

    // Reduce energy
    float frame_energy = warp_reduce_sum(shared_energy[tid]);

    if (tid == 0) {
        frame_energy = frame_energy / frame_length;
        vad[frame_idx] = (frame_energy > threshold) ? 1.0f : 0.0f;
    }
}

// Formant extraction kernel using LPC (Linear Predictive Coding)
__global__ void formant_extraction_kernel(float *spectrogram, float *formants, int n_frames, int n_freqs, int num_formants, float sample_rate) {
    int frame_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (frame_idx >= n_frames) return;
    
    // Simplified LPC on spectral data
    float lpc_coeffs[10]; // Assume order 10
    float energy = 0.0f;
    
    // Compute LPC coefficients (simplified)
    for (int i = 0; i < 10; i++) {
        lpc_coeffs[i] = 0.0f; // Placeholder for actual LPC computation
    }
    
    // Find formant peaks
    for (int f = 0; f < num_formants; f++) {
        float max_val = 0.0f;
        int max_idx = 0;
        for (int k = 100; k < n_freqs - 100; k += 50) { // Search for peaks
            if (spectrogram[frame_idx * n_freqs + k] > max_val) {
                max_val = spectrogram[frame_idx * n_freqs + k];
                max_idx = k;
            }
        }
        formants[frame_idx * num_formants + f] = (float)max_idx * sample_rate / (2.0f * n_freqs);
    }
}

// Vocoder synthesis kernel (simplified HiFi-GAN style)
__global__ void vocoder_synthesis_kernel(float *mel_spectrogram, float *audio_out, int n_frames, int n_mels, int hop_length) {
    int frame_idx = blockIdx.x;
    int tid = threadIdx.x;
    
    if (frame_idx >= n_frames) return;
    
    extern __shared__ float shared_mel[];
    
    // Load mel frame
    if (tid < n_mels) {
        shared_mel[tid] = mel_spectrogram[frame_idx * n_mels + tid];
    }
    __syncthreads();
    
    // Simplified Griffin-Lim style synthesis (placeholder for full vocoder)
    for (int t = 0; t < hop_length; t += blockDim.x) {
        if (tid + t < hop_length) {
            // Generate audio sample using inverse STFT approximation
            float phase = 0.0f; // Random phase or estimated
            float mag = 0.0f;
            for (int k = 0; k < n_mels; k++) {
                mag += shared_mel[k] * sinf(2.0f * PI * k * (tid + t) / (float)hop_length + phase);
            }
            audio_out[frame_idx * hop_length + tid + t] = mag / (float)n_mels;
        }
    }
}

// Windowing and packing kernel for STFT
__global__ void window_and_pack_kernel(float *audio, float *windowed, int n_samples, int n_fft, int hop_length) {
    int frame_idx = blockIdx.x;
    int tid = threadIdx.x;

    int frame_start = frame_idx * hop_length;

    // Apply Hann window
    if (tid < n_fft) {
        float window_val = 0.5f * (1.0f - cosf(2.0f * PI * tid / (n_fft - 1.0f))); // Hann window
        int audio_idx = frame_start + tid;
        float sample = (audio_idx < n_samples) ? audio[audio_idx] : 0.0f;
        windowed[frame_idx * n_fft + tid] = sample * window_val;
    }
}

// Magnitude computation kernel
__global__ void compute_magnitude_from_complex_kernel(hipfftComplex *complex_spec, float *magnitude, int n_frames, int n_fft) {
    int frame_idx = blockIdx.x;
    int tid = threadIdx.x;
    int n_bins = n_fft / 2 + 1;

    if (tid < n_bins) {
        int idx = frame_idx * n_bins + tid;
        // Correct indexing for R2C output: n_bins per frame
        hipfftComplex c = complex_spec[frame_idx * n_bins + tid];
        magnitude[idx] = sqrtf(c.x * c.x + c.y * c.y);
    }
}

// Host function to launch pitch detection (updated signature to match bindings)
void launch_pitch_detection(torch::Tensor& input, torch::Tensor& output, float sample_rate) {
    float *d_audio = input.data_ptr<float>();
    float *d_pitch = output.data_ptr<float>();
    int n_samples = input.size(0);
    int frame_length = 1024;
    int hop_length = 256;
    float fmin = 80.0f;
    float fmax = 400.0f;
    float threshold = 0.1f;

    int n_frames = std::max<int>(0, (n_samples - frame_length) / hop_length + 1);
    if (n_frames <= 0) {
        CUDA_CHECK(hipMemset(d_pitch, 0, output.numel() * sizeof(float)));
        return;
    }

    dim3 block(256);
    dim3 grid(n_frames);
    size_t shared_mem = frame_length * sizeof(float);
    pitch_detection_kernel<<<grid, block, shared_mem>>>(d_audio, d_pitch, n_samples, frame_length, hop_length, fmin, fmax, threshold, sample_rate);
    CUDA_CHECK(hipGetLastError());
}

// Host function for formant extraction (updated signature)
void launch_formant_extraction(torch::Tensor& input, torch::Tensor& output, float sample_rate) {
    float *d_spectrogram = input.data_ptr<float>();
    float *d_formants = output.data_ptr<float>();
    int n_frames = input.size(0);
    int n_freqs = input.size(1);
    int num_formants = output.size(1);

    int threads = 256;
    int blocks = (n_frames + threads - 1) / threads;
    formant_extraction_kernel<<<blocks, threads>>>(d_spectrogram, d_formants, n_frames, n_freqs, num_formants, sample_rate);
    CUDA_CHECK(hipGetLastError());
}

// Host function for vocoder synthesis (updated signature)
void launch_vocoder_synthesis(torch::Tensor& mel_spec, torch::Tensor& audio_out) {
    float *d_mel = mel_spec.data_ptr<float>();
    float *d_audio = audio_out.data_ptr<float>();
    int n_frames = mel_spec.size(0);
    int n_mels = mel_spec.size(1);
    int hop_length = 256;

    dim3 block(256);
    dim3 grid(n_frames);
    size_t shared_mem = n_mels * sizeof(float);
    vocoder_synthesis_kernel<<<grid, block, shared_mem>>>(d_mel, d_audio, n_frames, n_mels, hop_length);
    CUDA_CHECK(hipGetLastError());
}

// Host function for voice activity detection
void launch_voice_activity_detection(torch::Tensor& input, torch::Tensor& output, float threshold) {
    float *d_audio = input.data_ptr<float>();
    float *d_vad = output.data_ptr<float>();
    int n_samples = input.size(0);
    int frame_length = 1024;  // Default
    int hop_length = 256;     // Default

    int n_frames = std::max<int>(0, (n_samples - frame_length) / hop_length + 1);
    if (n_frames <= 0) {
        CUDA_CHECK(hipMemset(d_vad, 0, output.numel() * sizeof(float)));
        return;
    }

    dim3 block(256);
    dim3 grid(n_frames);
    size_t shared_mem = 256 * sizeof(float);
    vad_kernel<<<grid, block, shared_mem>>>(d_audio, d_vad, n_samples, frame_length, hop_length, threshold);
    CUDA_CHECK(hipGetLastError());
}

// Host function for spectrogram computation (matching bindings signature)
void launch_spectrogram_computation(torch::Tensor& input, torch::Tensor& output, int n_fft, int hop_length, int win_length) {
    float *d_audio = input.data_ptr<float>();
    float *d_spectrogram = output.data_ptr<float>();
    int n_samples = input.size(0);

    int n_frames = std::max<int>(0, (n_samples - win_length) / hop_length + 1);
    if (n_frames <= 0) {
        CUDA_CHECK(hipMemset(d_spectrogram, 0, output.numel() * sizeof(float)));
        return;
    }

    // Allocate temporary buffers
    float *d_windowed;
    hipfftComplex *d_fft_output;
    CUDA_CHECK(hipMalloc(&d_windowed, n_frames * n_fft * sizeof(float)));
    // R2C transform outputs (n_fft/2 + 1) complex numbers per frame
    CUDA_CHECK(hipMalloc(&d_fft_output, n_frames * (n_fft/2 + 1) * sizeof(hipfftComplex)));

    // Step 1: Apply windowing and pack frames
    dim3 block(256);
    dim3 grid(n_frames);
    window_and_pack_kernel<<<grid, block>>>(d_audio, d_windowed, n_samples, n_fft, hop_length);
    CUDA_CHECK(hipGetLastError());

    // Step 2: Execute cuFFT forward transform
    execute_cufft_forward(d_windowed, d_fft_output, n_frames, n_fft);

    // Step 3: Compute magnitude spectrogram
    int n_bins = n_fft / 2 + 1;
    // Make sure we have enough threads to process all bins
    dim3 mag_block(256);
    dim3 mag_grid(n_frames);
    compute_magnitude_from_complex_kernel<<<mag_grid, mag_block>>>(d_fft_output, d_spectrogram, n_frames, n_fft);
    CUDA_CHECK(hipGetLastError());

    // Cleanup
    CUDA_CHECK(hipFree(d_windowed));
    CUDA_CHECK(hipFree(d_fft_output));
}